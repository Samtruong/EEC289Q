
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <set>
#include <sstream>
#include <string>
#include <fstream>
#include <iostream>
#include <cstring>
#include <queue>

using namespace std;

//#include <device_vector.h>

  //cudaMallocManaged(& bins, numC*numV*sizeof(int));

/*
__global__
void makeBins(int * coloredGraph, int numV, int numC, std::queue<int>* bins)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;
  for(int i = index ; i < numV; i+= stride)
  {
    bins[coloredGraph[i]].push(i);
  }
}
*/

struct AdjListNode {
	int vertex;
	struct AdjListNode * next;
};

struct AdjList {
	struct AdjListNode * head;
};

struct AdjListNode* newAdjListNode(int vertex)
{
	struct AdjListNode * newNode = new AdjListNode;
	newNode -> vertex = vertex;
	newNode -> next = NULL;
	return newNode;
};

struct AdjList*  populateList(bool *graph, int numV)
{
	struct AdjList * list =  new AdjList[numV ];
	struct AdjListNode * node;
	for (int i = 0; i < numV; i++)
	{
		node = newAdjListNode(i + 1);
		node -> next = NULL;
		list[i].head = node;
		
	}
	//struct AdjList list[numV];
	for (int i = 0; i < numV; i++)
	{
		for (int j = 0; j < numV ; j++)
		{
			if (graph[i*numV + j])
			{
				//cout << i + 1 << " and " << j+1 << " are connected. " << endl;
				struct AdjListNode * toAdd = newAdjListNode(j + 1);
				toAdd -> next = NULL;
				struct AdjListNode * newNode = list[i].head;
				while (newNode -> next)
					newNode = newNode -> next;
				newNode -> next = toAdd;

			}

		}

	}
	return list;
}

void printList (struct AdjList * list, int numV)
{
	for (int i = 0; i < numV; i++)
	{
		struct AdjListNode* pCrawl = list[i].head;
        printf("\n Adjacency list of vertex %d\n head ", i + 1);
        while (pCrawl) {
            printf("-> %d", pCrawl->vertex);
            pCrawl = pCrawl->next;
        }
        printf("\n");
	}
}
/*
__global__
void reduceColors (bool *graph, int *coloredGraph, int numV, int numC, int numIterations, struct AdjList *list)
{
	for (int i = 0; i < numIterations; i++)
	{
		int vertex1 = rand();
		int vertex2 = rand();
		int vertex1Color = coloredGraph[vertex1];
		struct AdjListNode *ptr = list[vertex2].head;
		if (coloredGraph[vertex1] == coloredGraph[vertex2])
			continue;
		if (!graph[vertex1*numV + vertex2])
		{
			//loop through all adjacent vertices of vertex 2 to determine if same color exists.
			while (ptr -> next)
			{
				ptr = ptr -> next;
				if (coloredGraph[ptr -> vertex] == vertex1Color) continue;
			}	
			if (coloredGraph[vertex1] < coloredGraph[vertex2])
				coloredGraph[vertex2] = coloredGraph[vertex1];
			else
				coloredGraph[vertex1] = coloredGraph[vertex2];
		}
	}
}
*/

/*
__global__
void reduceBins(int * coloredGraph, int numV, int numC, std::queue<int> * bins, int threshold)
{
	int binIndex = blockIdx.x % numC;
	std::queue<int> bin = bins[binIndex];
	int iteration = 0;
	while (iteration < threshold)
	{	
		int currentVertex = queue.pop();
			

	}	

}
*/

void ReadColFile(const char filename[], bool** graph, int* V)
{
   string line;
   ifstream infile(filename);
   if (infile.fail()) {
      printf("Failed to open %s\n", filename);
      return;
   }

   int num_rows, num_edges;

   while (getline(infile, line)) {
      istringstream iss(line);
      string s;
      int node1, node2;
      iss >> s;
      if (s == "p") {
         iss >> s; // read string "edge"
         iss >> num_rows;
         iss >> num_edges;
         *V = num_rows;
         *graph = new bool[num_rows * num_rows];
         memset(*graph, 0, num_rows * num_rows * sizeof(bool));
         continue;
      } else if (s != "e")
         continue;

      iss >> node1 >> node2;

      // Assume node numbering starts at 1
      (*graph)[(node1 - 1) * num_rows + (node2 - 1)] = true;
      (*graph)[(node2 - 1) * num_rows + (node1 - 1)] = true;
   }
   infile.close();
}



int main(int argc, char *argv[])
{

  bool *graph;
  int V;
  int *color;

   if (string(argv[1]).find(".col") != string::npos)
      ReadColFile(argv[1], &graph, &V);
  else
  	return -1;

  AdjList * list = populateList(graph, V);
  printList(list, V);
//Code to make random graph
/*
  const int numV = 10;
  const int numC = 5;
  int coloredGraph[numV];

  for(int i = 0; i <  numV; i++)
  {
    coloredGraph[i] = rand() % 1000 + 1;
  }
*/


  
  //std::queue<int>  bins [numC];

  //makeBins<<<1,1>>>(coloredGraph, numV, numC, bins);
  
//cudaFree(bins);  
return 0;
}
