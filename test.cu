
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <set>
#include <sstream>
#include <string>
#include <fstream>
#include <iostream>
#include <cstring>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

using namespace std;

//#include <device_vector.h>

  //cudaMallocManaged(& bins, numC*numV*sizeof(int));

/*
__global__
void makeBins(int * coloredGraph, int numV, int numC, std::queue<int>* bins)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;
  for(int i = index ; i < numV; i+= stride)
  {
    bins[coloredGraph[i]].push(i);
  }
}
*/

struct AdjListNode {
	int vertex;
	struct AdjListNode * next;
};

struct AdjList {
	struct AdjListNode * head;
};

struct AdjListNode* newAdjListNode(int vertex)
{
	struct AdjListNode * newNode = new AdjListNode;
	newNode -> vertex = vertex;
	newNode -> next = NULL;
	return newNode;
};

struct AdjList*  populateList(bool *graph, int numV)
{
	struct AdjList * list =  new AdjList[numV ];
	struct AdjListNode * node;
	for (int i = 0; i < numV; i++)
	{
		node = newAdjListNode(i + 1);
		node -> next = NULL;
		list[i].head = node;
		
	}
	//struct AdjList list[numV];
	for (int i = 0; i < numV; i++)
	{
		for (int j = 0; j < numV ; j++)
		{
			if (graph[i*numV + j])
			{
				//cout << i + 1 << " and " << j+1 << " are connected. " << endl;
				struct AdjListNode * toAdd = newAdjListNode(j + 1);
				toAdd -> next = NULL;
				struct AdjListNode * newNode = list[i].head;
				while (newNode -> next)
					newNode = newNode -> next;
				newNode -> next = toAdd;

			}

		}

	}
	return list;
}

void printList (struct AdjList * list, int numV)
{
	for (int i = 0; i < numV; i++)
	{
		struct AdjListNode* pCrawl = list[i].head;
        printf("\n Adjacency list of vertex %d\n head ", i + 1);
        while (pCrawl) {
            printf("-> %d", pCrawl->vertex);
            pCrawl = pCrawl->next;
        }
        printf("\n");
	}
}

__global__
void reduceColors (bool *graph, int *coloredGraph, int numV, int numIterations, struct AdjList *list, unsigned int seed)
{
	for (int i = 0; i < numIterations; i++)
	{
		hiprandState_t state;
		hiprand_init(seed, 0, 1, &state);

		int vertex1 = 0; //curand(&state) % numV;
		int vertex2 = 2; //curand(&state) % numV;
		printf("vertex1 %i vertex 2 %i     \n", vertex1, vertex2);	
		printf("hello\n");
		//int vertex1 = rand();
		//int vertex2 = rand();
		int vertex1Color = coloredGraph[vertex1];
		printf("vertex 1 color %i   \n", vertex1Color);

		if (coloredGraph[vertex1] == coloredGraph[vertex2])
		{
			printf("colors equal\n");
			continue;
		}
			printf("vertex 1 color %i   \n", vertex1Color);
		

		if (!graph[vertex1*numV + vertex2])
		{
			printf("changing color \n");
			struct AdjListNode *ptr = list[vertex2].head;
			//loop through all adjacent vertices of vertex 2 to determine if same color exists.
			while (ptr -> next)
			{
				ptr = ptr -> next;
				if (coloredGraph[ptr -> vertex] == vertex1Color) continue;
			}	
			if (coloredGraph[vertex1] < coloredGraph[vertex2])
				coloredGraph[vertex2] = coloredGraph[vertex1];
			else
				coloredGraph[vertex1] = coloredGraph[vertex2];
		}
	}

}


/*
__global__
void reduceBins(int * coloredGraph, int numV, int numC, std::queue<int> * bins, int threshold)
{
	int binIndex = blockIdx.x % numC;
	std::queue<int> bin = bins[binIndex];
	int iteration = 0;
	while (iteration < threshold)
	{	
		int currentVertex = queue.pop();
			

	}	

}
*/

void ReadColFile(const char filename[], bool** graph, int* V)
{
   string line;
   ifstream infile(filename);
   if (infile.fail()) {
      printf("Failed to open %s\n", filename);
      return;
   }

   int num_rows, num_edges;

   while (getline(infile, line)) 
   {
      istringstream iss(line);
      string s;
      int node1, node2;
      iss >> s;
      if (s == "p") {
         iss >> s; // read string "edge"
         iss >> num_rows;
         iss >> num_edges;
         *V = num_rows;
         *graph = new bool[num_rows * num_rows];
         memset(*graph, 0, num_rows * num_rows * sizeof(bool));
         continue;
      } else if (s != "e")
         continue;

      iss >> node1 >> node2;

      // Assume node numbering starts at 1
      (*graph)[(node1 - 1) * num_rows + (node2 - 1)] = true;
      (*graph)[(node2 - 1) * num_rows + (node1 - 1)] = true;
   }
   infile.close();
}

void trivialColor(int * color, int V)
{
	for (int i = 0; i < V; i++)
	{
		color[i] = i;
	}
}

int main(int argc, char *argv[])
{

  bool *graph;
  int V;
  int *color;

  

   if (string(argv[1]).find(".col") != string::npos)
      ReadColFile(argv[1], &graph, &V);
  else
  	return -1;
  color = new int[V];

  AdjList * list = populateList(graph, V);
  printList(list, V);
  trivialColor(color, V);
  reduceColors<<<1, 1>>>(graph, color, V, 1, list, time(NULL));
  
  hipDeviceSynchronize();
 for(int i = 0; i <  V; i++)
  {
    cout << i << "  " << color[i] << endl;
   }

//Code to make random graph
/*
  const int numV = 10;
  const int numC = 5;
  int coloredGraph[numV];

 
*/


  
  //std::queue<int>  bins [numC];

  //makeBins<<<1,1>>>(coloredGraph, numV, numC, bins);
  
//cudaFree(bins);  
return 0;
}
