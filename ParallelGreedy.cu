#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <set>
#include <sstream>
#include <string>
#include <fstream>
#include <iostream>
#include <cstring>
#include <thrust/scan.h>
#include <thrust/execution_policy.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

using namespace std;


__global__ void CopyGraph(int* h_graph, int* pre_graph, int length)
{
  int index = threadIdx.x + blockDim.x * blockIdx.x;
  int stride = blockDim.x * gridDim.x;
  for(int i = index; i < length; i += stride){h_graph[i] = pre_graph[i];}
}

__global__ void GraphGenerator(int* matrix, int* dimension, int* address, int V, int *h_graph)
{
  int index = threadIdx.x + blockDim.x * blockIdx.x;
  int stride = blockDim.x * gridDim.x;
  for(int i = index; i < V; i += stride)
  {
    for (int j = 0; j < V; j++)
    {
      if(matrix[i*V + j])
      {
        dimension[i]++;
      }
    }
  }
  __syncthreads();
  thrust::exclusive_scan(thrust::device,&dimension[0],&dimension[V], &address[0]);
  for(int i = index; i < V; i += stride)
  {
    int a = address[i];
    int j = 0;
    for (int k = 0; k < V; k++)
    {
      if (matrix[i*V + k])
      {
        h_graph[a + j] = k;
        j++;
      }
    }
  }
}

__global__ void PermutationGenerator(int V, int*result, int numVersion, int shuffle_degree)
{
  unsigned long long seed = blockDim.x;
  unsigned long long sequence = threadIdx.x;
  unsigned long long offset = 0;
  hiprandState_t state;
  hiprand_init(seed,sequence,offset,&state);
  int index = threadIdx.x + blockDim.x * blockIdx.x;
  int stride = blockDim.x * gridDim.x;
  int num1,num2,holder;
  for(int i = index; i < V*numVersion; i+= stride){result[i] = i%V;}
  __syncthreads();
  for(int j = index; j<numVersion; j+=stride)
  {
    for(int k = 0; k < shuffle_degree; k++)
    {
      num1 = j*V + hiprand(&state) % V;
      num2 = j*V + hiprand(&state) % V;
      holder = result[num1];
      result[num1] = result[num2];
      result[num2] = holder;
    }
  }
}

__device__ void Color(int* h_graph, int startingAddress,int curVertex, int a, int d, int* result)
{
  //int result[curVertex] = 1;
  int color = 1;
  // printf("in color on vertex %i\n", startingAddress);
  // //printf("h_graph\n");
  // for (int i = 0; i < d; i++)
  //   printf("%i ", h_graph[a+i]);
  // printf("\n");
  // printf("dimension %i\n", d);
  // printf("address %i\n", a);
  for (int i = 0; i < d; i++)
  {
    // printf ("hgraph referecned: %i at %i\n", result[h_graph[a + i]], h_graph[a + i] );
    if (color == result[startingAddress + h_graph[a + i]])
    {
      // printf("color incremented\n");
      i = 0;
      color ++;
      continue;
    }
  }
  // printf("curVertex %i\n", curVertex);
  result[startingAddress +curVertex] = color;

}
__global__ void RandomizedParallelGreedy(int* h_graph, int* dimension,
                 int* address, int* sequence,int V, int numVersion, int* result)
{
  // printf("Sequence:\n");
  // for (int i = 0; i < V *numVersion; i++)
  // {
  //   printf("%i", sequence[i]);
  // }
  // printf("1\n");
  int index = threadIdx.x + blockDim.x * blockIdx.x;
  int stride = blockDim.x * gridDim.x;
  int a,d;

  // extern __shared__ int d_graph[];
  // extern __shared__ int d_dimension[];
  // extern __shared__ int d_address[];
  /*
  extern __shared__ int d_dimension[];
  extern __shared__ int d_address[];
  */
// printf("2\n");
  int length = dimension[V - 1] + address[V - 1]; //length of h_graph;

  //copy to shared memory:

  //for(int i = index; i < length; i+= stride){d_graph[i] = h_graph[i];}
  __syncthreads();
// printf("3\n");

  /*for(int i = index; i < V; i+= stride)
  {
    d_dimension[i] = dimension[i];
    d_address[i] = address[i];
  }*/
  __syncthreads();
  //end copy to shared memory
  // printf("4\n");

  for(int j = index; j < numVersion; j +=stride)
  {
    for(int k = 0; k < V; k++)
    {
      int curVertex = sequence[j*V+k];
      a = address[curVertex]; //address of first neighboor
      d = dimension[curVertex];//number of neighboor
      Color(h_graph,j*V,curVertex, a, d, result);
      __syncthreads();
    }
    // printf("nextVersion\n");
  }
}
//================================Utility Functions=======================================
__host__ __device__
int CountColors(int V, int* color)
{
   int num_colors = 0;
   set<int> seen_colors;

   for (int i = 0; i < V; i++) {
      if (seen_colors.find(color[i]) == seen_colors.end()) {
         seen_colors.insert(color[i]);
         num_colors++;
      }
   }

   return num_colors;
}

//Load raw .co data
void getDimension(const char filename[], int* V)
{
   string line;
   ifstream infile(filename);
   if (infile.fail()) {
      printf("Failed to open %s\n", filename);
      return;
   }

   int num_rows;

   while (getline(infile, line))
   {
      istringstream iss(line);
      string s;
      iss >> s;
      if (s == "p") {
         iss >> s; // read string "edge"
         iss >> num_rows;
         *V = num_rows;
         break;
      }
   }
   infile.close();
}

void ReadColFile(const char filename[], int* graph, int V)
{
   string line;
   ifstream infile(filename);
   if (infile.fail()) {
      printf("Failed to open %s\n", filename);
      return;
   }

   while (getline(infile, line)) {
      istringstream iss(line);
      string s;
      int node1, node2;
      iss >> s;
      if (s != "e")
         continue;

      iss >> node1 >> node2;

      // Assume node numbering starts at 1
      (graph)[(node1 - 1) * V + (node2 - 1)] = 1;
      (graph)[(node2 - 1) * V + (node1 - 1)] = 1;
   }
   infile.close();
}

//print graph Matrix
void PrintMatrix(int* matrix, int M, int N) {
   for (int row=0; row<M; row++)
   {
      for(int columns=0; columns<N; columns++)
      {
         printf("%i", matrix[row * N + columns]);
      }
      printf("\n");
   }
}


//===================================Main=======================================

int main(int argc, char* argv[])
{
   int * matrix;
   int * pre_graph;
   int * h_graph;
   int * sequence;
   int * dimension;
   int * address;
   int * result; //Added
   int V;
   int numVersion;

   numVersion = 100;


   if (string(argv[1]).find(".col") != string::npos)
   {
     getDimension(argv[1], &V);
     hipMallocManaged(&matrix,sizeof(int)*V*V);
     ReadColFile(argv[1],matrix,V);
   }

   //else if (string(argv[1]).find(".mm") != string::npos)
      //ReadMMFile(argv[1], &graph, &V);
   else
      return -1;
   hipMallocManaged(&sequence, sizeof(int) * V * numVersion);
   hipMallocManaged(&dimension,sizeof(int)*V);
   hipMallocManaged(&address,sizeof(int)*V);
   hipMallocManaged(&result, sizeof(int) *V*numVersion);
   hipMallocManaged(&pre_graph,sizeof(int)*V*V);

   //Added for testing
   //h_graph 2, 3, 1, 3, 4,1,2,4,2,3
   //dimension 2,3,3,2
   //address 0,2,5,8
   // hipMallocManaged(&h_graph,sizeof(int)*V*V);
   // h_graph[0]=1; h_graph[1]= 2; h_graph[2]=0; h_graph[3]=2; h_graph[4]=3;
   // h_graph[5]=0; h_graph[6]=1; h_graph[7]=3; h_graph[8]=1; h_graph[9]=2;
   //
   // dimension[0]=2; dimension[1]=3; dimension[2] = 3; dimension[3]=2;
   // address[0]=0; address[1]=2; address[2]=5; address[3]=8;

   GraphGenerator<<<256,1024>>>(matrix,dimension,address,V,pre_graph);
   hipDeviceSynchronize();
   hipMallocManaged(&h_graph,sizeof(int)* (dimension[V-1]+address[V-1]));
   CopyGraph<<<256,1024>>>(h_graph,pre_graph,dimension[V-1]+address[V-1]);
   hipDeviceSynchronize();

   PermutationGenerator<<<256,1024>>>(V,sequence,numVersion,V);
   hipDeviceSynchronize();

   RandomizedParallelGreedy<<<256,1024>>>
   (h_graph, dimension, address, sequence, V, numVersion, result);
   hipDeviceSynchronize();


   printf("dimensions\n");
   for (int i = 0; i < V; i++)
   {
     cout << dimension[i] << " ";
   }
   cout << endl;
   printf("address\n");
   for (int i = 0; i < V; i++)
   {
     cout << address[i] << " ";
   } cout << endl;
   for (int i =0; i < (dimension[V-1]+address[V-1]); i++){printf("%i ", h_graph[i]);}

   cout<<endl;
   printf("coloring:\n");
   for (int i = 0; i < V*numVersion; i++)
   {
    cout << result[i] << " ";
    if(i%V == V-1){cout<<endl;}
   }

   hipFree(h_graph);
   hipFree(dimension);
   hipFree(sequence);
   hipFree(address);
   hipFree(result);
   hipFree(matrix);
   hipFree(pre_graph);
   return 0;
}
