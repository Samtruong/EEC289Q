#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <set>
#include <sstream>
#include <string>
#include <fstream>
#include <iostream>
#include <cstring>
#include <thrust/scan.h>
#include <thrust/execution_policy.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

using namespace std;

void SerialThrust(int* h_graph, int* dimension, int V)
{
  for(int row = 0; row < V; row++)
  {
    thrust::exclusive_scan(&h_graph[V*row],&h_graph[V*row + V],&h_graph[V*row]);
    dimension[row] = h_graph[V*row + V -1]+1;
  }
}

__global__ void ParallelThrust(int* h_graph, int* dimension, int V)
{
  int index = threadIdx.x + blockDim.x * blockIdx.x;
  int stride = blockDim.x * gridDim.x;
  for(int i = index; index < V*V; index += stride)
  {
    thrust::exclusive_scan(thrust::device,&h_graph[i*V],&h_graph[i*V+V],&h_graph[i*V]);
    dimension[i] =  h_graph[V*i + V -1]+1;
  }
}

__global__ void PermutationGenerator(int V, int*result, int numVersion, int shuffle_degree)
{
  unsigned long long seed = blockDim.x;
  unsigned long long sequence = threadIdx.x;
  unsigned long long offset = 0;
  hiprandState_t state;
  hiprand_init(seed,sequence,offset,&state);
  int index = threadIdx.x + blockDim.x * blockIdx.x;
  int stride = blockDim.x * gridDim.x;
  int num1,num2,holder;
  for(int i = index; i < V*numVersion; i+= stride){result[i] = i%V;}
  __syncthreads();
  for(int j = index; j<numVersion; j+=stride)
  {
    for(int k = 0; k < shuffle_degree; k++)
    {
      num1 = j*V + hiprand(&state) % V;
      num2 = j*V + hiprand(&state) % V;
      holder = result[num1];
      result[num1] = result[num2];
      result[num2] = holder;
    }
  }
}
__global__ void RandomizedParallelGreedy(int* h_graph, int* dimension,
                       int* address, int* sequence,int V, int* result)
{
  int index = threadIdx.x + blockDim.x * blockIdx.x;
  int stride = blockDim.x * gridDim.x;
  extern int sequence[];

  __shared__ int* d_graph;
  __shared__ int* d_dimension;
  __shared__ int* d_address;

  int length = dimension[V - 1] + address[V - 1]; //length of h_graph;

  //copy to shared memory:
  for(int i = index; i < length; i+= stride){d_graph[i] = h_graph[i];}
  __syncthreads();

  for(int i = index; i < V; i+= stride)
  {
    d_dimension[i] = dimension[i];
    d_address[i] = address[i];
  }
  __syncthreads();
  //end copy to shared memory

  for(int j = index, j < )
}
//================================Utility Functions=======================================

//Load raw .co data
void getDimension(const char filename[], int* V)
{
   string line;
   ifstream infile(filename);
   if (infile.fail()) {
      printf("Failed to open %s\n", filename);
      return;
   }

   int num_rows;

   while (getline(infile, line))
   {
      istringstream iss(line);
      string s;
      iss >> s;
      if (s == "p") {
         iss >> s; // read string "edge"
         iss >> num_rows;
         *V = num_rows;
         break;
      }
   }
   infile.close();
}

void ReadColFile(const char filename[], int* graph, int V)
{
   string line;
   ifstream infile(filename);
   if (infile.fail()) {
      printf("Failed to open %s\n", filename);
      return;
   }

   while (getline(infile, line)) {
      istringstream iss(line);
      string s;
      int node1, node2;
      iss >> s;
      if (s != "e")
         continue;

      iss >> node1 >> node2;

      // Assume node numbering starts at 1
      (graph)[(node1 - 1) * V + (node2 - 1)] = 1;
      (graph)[(node2 - 1) * V + (node1 - 1)] = 1;
   }
   infile.close();
}

//print graph Matrix
void PrintMatrix(int* matrix, int M, int N) {
   for (int row=0; row<M; row++)
   {
      for(int columns=0; columns<N; columns++)
      {
         printf("%i", matrix[row * N + columns]);
      }
      printf("\n");
   }
}


//===================================Main=======================================

int main(int argc, char* argv[])
{
   int* h_graph,sequence,dimension;
   int V,numVersion;
   numversion = 100;
   //int* color;
   if (string(argv[1]).find(".col") != string::npos)
   {
     getDimension(argv[1], &V);
     hipMallocManaged(&h_graph,sizeof(int)*V*V);
     ReadColFile(argv[1],h_graph,V);
   }
   //else if (string(argv[1]).find(".mm") != string::npos)
      //ReadMMFile(argv[1], &graph, &V);
   else
      return -1;

   hipMallocManaged(&sequence,sizeof(int)*V*numVersion);
   hipMallocManaged(&dimension,sizeof(int)*V);

   ParallelThrust<<<V,V>>>(h_graph,dimension,V);
   hipDeviceSynchronize();
   PermutationGenerator<<<256,1024>>>(V,sequence,numVersion,V)
   hipDeviceSynchronize();
   RandomizedParallelGreedy<<<>>>();
   hipDeviceSynchronize();
   // cout<<"Scan Graph"<<endl;
   // PrintMatrix(h_graph,V,V);
   // cout<<"dimension"<<endl;
   // PrintMatrix(dimension,1,V);
   hipFree(h_graph);
   hipFree(dimension);
   hipFree(sequence);
   return 0;
}
