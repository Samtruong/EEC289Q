#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <set>
#include <sstream>
#include <string>
#include <fstream>
#include <iostream>
#include <cstring>
#include <thrust/scan.h>
#include <thrust/execution_policy.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

using namespace std;

void SerialThrust(int* h_graph, int* dimension, int V)
{
  for(int row = 0; row < V; row++)
  {
    thrust::exclusive_scan(&h_graph[V*row],&h_graph[V*row + V],&h_graph[V*row]);
    dimension[row] = h_graph[V*row + V -1]+1;
  }
}

__global__ void ParallelThrust(int* h_graph, int* dimension, int V)
{
  int index = threadIdx.x + blockDim.x * blockIdx.x;
  int stride = blockDim.x * gridDim.x;
  for(int i = index; index < V*V; index += stride)
  {
    thrust::exclusive_scan(thrust::device,&h_graph[i*V],&h_graph[i*V+V],&h_graph[i*V]);
    dimension[i] =  h_graph[V*i + V -1]+1;
  }
}

__global__ void PermutationGenerator(int V, int*result, int numVersion, int shuffle_degree)
{
  unsigned long long seed = blockDim.x;
  unsigned long long sequence = threadIdx.x;
  unsigned long long offset = 0;
  hiprandState_t state;
  hiprand_init(seed,sequence,offset,&state);
  int index = threadIdx.x + blockDim.x * blockIdx.x;
  int stride = blockDim.x * gridDim.x;
  int num1,num2,holder;
  for(int i = index; i < V*numVersion; i+= stride){result[i] = i%V;}
  __syncthreads();
  for(int j = index; j<numVersion; j+=stride)
  {
    for(int k = 0; k < shuffle_degree; k++)
    {
      num1 = j*V + hiprand(&state) % V;
      num2 = j*V + hiprand(&state) % V;
      holder = result[num1];
      result[num1] = result[num2];
      result[num2] = holder;
    }
  }
}

__device__ void Color(int* h_graph, int startingAddress,int curVertex, int a, int d, int* result)
{
  //int result[curVertex] = 1;
  int color = 1;
  printf("in color on vertex %i\n", startingAddress);
  //printf("h_graph\n");
  for (int i = 0; i < d; i++)
    printf("%i ", h_graph[a+i]);
  printf("\n");
  printf("dimension %i\n", d);
  printf("address %i\n", a);
  for (int i = 0; i < d; i++)
  {
    // printf ("hgraph referecned: %i at %i\n", result[h_graph[a + i]], h_graph[a + i] );
    if (color == result[startingAddress + h_graph[a + i]])
    {
      // printf("color incremented\n");
      i = 0;
      color ++;
      continue;
    }
  }
  // printf("curVertex %i\n", curVertex);
  result[startingAddress +curVertex] = color;

}
__global__ void RandomizedParallelGreedy(int* h_graph, int* dimension,
                 int* address, int* sequence,int V, int numVersion, int* result)
{
  // printf("Sequence:\n");
  // for (int i = 0; i < V *numVersion; i++)
  // {
  //   printf("%i", sequence[i]);
  // }
  // printf("1\n");
  int index = threadIdx.x + blockDim.x * blockIdx.x;
  int stride = blockDim.x * gridDim.x;
  int a,d;

  extern __shared__ int d_graph[];
  extern __shared__ int d_dimension[];
  extern __shared__ int d_address[];
  /*
  extern __shared__ int d_dimension[];
  extern __shared__ int d_address[];
  */
// printf("2\n");
  int length = dimension[V - 1] + address[V - 1]; //length of h_graph;

  //copy to shared memory:

  for(int i = index; i < length; i+= stride){d_graph[i] = h_graph[i];}
  __syncthreads();
// printf("3\n");

  /*for(int i = index; i < V; i+= stride)
  {
    d_dimension[i] = dimension[i];
    d_address[i] = address[i];
  }*/
  __syncthreads();
  //end copy to shared memory
  // printf("4\n");

  for(int j = index; j < numVersion; j +=stride)
  {
    for(int k = 0; k < V; k++)
    {
      int curVertex = sequence[j*V+k];
      a = address[curVertex]; //address of first neighboor
      d = dimension[curVertex];//number of neighboor
      Color(h_graph,j*V,curVertex, a, d, result);
      __syncthreads();
    }
    // printf("nextVersion\n");
  }
}
//================================Utility Functions=======================================

//Load raw .co data
void getDimension(const char filename[], int* V)
{
   string line;
   ifstream infile(filename);
   if (infile.fail()) {
      printf("Failed to open %s\n", filename);
      return;
   }

   int num_rows;

   while (getline(infile, line))
   {
      istringstream iss(line);
      string s;
      iss >> s;
      if (s == "p") {
         iss >> s; // read string "edge"
         iss >> num_rows;
         *V = num_rows;
         break;
      }
   }
   infile.close();
}

void ReadColFile(const char filename[], int* graph, int V)
{
   string line;
   ifstream infile(filename);
   if (infile.fail()) {
      printf("Failed to open %s\n", filename);
      return;
   }

   while (getline(infile, line)) {
      istringstream iss(line);
      string s;
      int node1, node2;
      iss >> s;
      if (s != "e")
         continue;

      iss >> node1 >> node2;

      // Assume node numbering starts at 1
      (graph)[(node1 - 1) * V + (node2 - 1)] = 1;
      (graph)[(node2 - 1) * V + (node1 - 1)] = 1;
   }
   infile.close();
}

//print graph Matrix
void PrintMatrix(int* matrix, int M, int N) {
   for (int row=0; row<M; row++)
   {
      for(int columns=0; columns<N; columns++)
      {
         printf("%i", matrix[row * N + columns]);
      }
      printf("\n");
   }
}


//===================================Main=======================================

int main(int argc, char* argv[])
{
   int* h_graph;
   int * sequence;
   int * dimension;
   int * address;
   int * result; //Added
   int V,numVersion;

   numVersion = 10;
   V = 4;

/*
   if (string(argv[1]).find(".col") != string::npos)
   {
     getDimension(argv[1], &V);
     hipMallocManaged(&h_graph,sizeof(int)*V*V);
     ReadColFile(argv[1],h_graph,V);
   }
   //else if (string(argv[1]).find(".mm") != string::npos)
      //ReadMMFile(argv[1], &graph, &V);
   else
      return -1;*/

  hipMallocManaged(&sequence, sizeof(int) * V * numVersion);
   hipMallocManaged(&dimension,sizeof(int)*V);
   hipMallocManaged(&address,sizeof(int)*V);
   hipMallocManaged(&result, sizeof(int) *V*numVersion);

   //Added for testing
   //h_graph 2, 3, 1, 3, 4,1,2,4,2,3
   //dimension 2,3,3,2
   //address 0,2,5,8
        hipMallocManaged(&h_graph,sizeof(int)*V*V);
   h_graph[0]=1; h_graph[1]= 2; h_graph[2]=0; h_graph[3]=2; h_graph[4]=3;
   h_graph[5]=0; h_graph[6]=1; h_graph[7]=3; h_graph[8]=1; h_graph[9]=2;

   dimension[0]=2; dimension[1]=3; dimension[2] = 3; dimension[3]=2;
   address[0]=0; address[1]=2; address[2]=5; address[3]=8;

// ParallelThrust<<<V,V>>>(h_graph,dimension,V);
   hipDeviceSynchronize();

   thrust::exclusive_scan(&dimension[0],&dimension[V], address);
   PermutationGenerator<<<256,1024>>>(V,sequence,numVersion,V);
   hipDeviceSynchronize();
   // printf("sequence:\n");
   // for (int i = 0; i < V*numVersion; i++)
   // {
   //  cout << sequence[i] << " ";
   //  if(i%4 == 3){cout<<endl;}
   // }

   RandomizedParallelGreedy<<<1,1, sizeof(h_graph)+sizeof(dimension)+sizeof(address)>>>
   (h_graph, dimension, address, sequence, V, numVersion, result);
   hipDeviceSynchronize();

   printf("coloring:\n");
   for (int i = 0; i < V*numVersion; i++)
   {
    cout << result[i] << " ";
    if(i%4 == 3){cout<<endl;}
   }

   hipFree(h_graph);
   hipFree(dimension);
   hipFree(sequence);
   return 0;
}
